
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>

// Function to be run on GPU
__global__ void vector_vector(int *a,int *b, int *c, int n){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //checking if thread id is less than n
  if(tid<n){
    c[tid] = a[tid] * b[tid];
  }

}

void init(int * arr, int n, int num){
    for(int i=0;i<n;i++){
        arr[i] = num;
    }
}

bool check(int *a, int *b, int *c, int n){

    for(int i=0;i<n;i++){
        if(c[i] != a[i]*b[i])
        return false;
    }

    return true;

}
int main(int argc, char * argv[]){

    //size of arr;
    int n = 100;

    int *a;
    int *b;
    int *c;

    //calculating the size of arr
    size_t bytes = n * sizeof(int);

    hipMallocManaged(&a,bytes);
    hipMallocManaged(&b,bytes);
    hipMallocManaged(&c,bytes);

    init(a,n,7);
    init(b,n,8);

    size_t threads = 10;
    size_t blocks = n/threads;

    // calling the kernel
    vector_vector<<<threads,blocks>>>(a,b,c,n);

    hipDeviceSynchronize();

    if(check(a,b,c,n)){
        printf("The output is correct");
    }else{
        printf("Incorrect Output");
    }

    return 0;

}